#include <stdio.h>
#include <hip/hip_runtime.h>


#include <iostream>
#include <string>
using namespace std;

#define SIZE 1280

__constant__ unsigned int gKey[8];
__constant__ short int gTable[8][16];

__device__ void F(unsigned int *itsR)
{
	unsigned int r[8];
	unsigned int mask = 0xF;
	for(int i = 0; i < 8; i ++) {
		r[i] = (*itsR & mask) >> 4 * i;
		mask <<= 4;
	}
	*itsR = 0;
	for(int i = 7; i >= 0; i --) {
		(*itsR) <<= 4;
		r[i] = gTable[i][r[i]];
		(*itsR) += r[i];
	}
	
}

__global__ void gostEncrypt(unsigned long long *data, unsigned long long *result) 
{

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int L = ((data[index] & 0xFFFFFFFF00000000) >> 32);
	unsigned int R = (data[index] & 0x00000000FFFFFFFF);
	const long long number = 4294967296L;
	int j = 0;
	for(int i = 1; i <= 32; i ++) {
	    unsigned int V = R;
	    if(i < 25)
			j = (i - 1) % 8;
		else
			j = (32 - i) % 8;
		long long buf = R + gKey[j];
		while(buf >= number)
			buf = buf - number;
		R = buf;
		F(&R);
		unsigned long long mask = 0x80000000;
		unsigned int leftBit;
		for(int k = 0; k < 11; k ++) {
			leftBit = R & mask;
			R <<= 1;
			if(leftBit != 0)
				R += 1;
		}
		R ^= L;
		L = V;
	}
	unsigned long long res = (unsigned long long) L << 32;
	res += R;
	result[index] = res;
}

__global__ void gostDeciphered(unsigned long long *data, unsigned long long *result) 
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int L = ((data[index] & 0xFFFFFFFF00000000) >> 32);
	unsigned int R = (data[index] & 0x00000000FFFFFFFF);
	const long long number = 4294967296L;
	int j = 0;
	for(int i = 1; i <= 32; i ++) {
	    unsigned int V = L;
	    if(i <= 8)
			j = (i - 1) % 8;
		else
			j = (32 - i) % 8;
		long long buf = L + gKey[j];
		while(buf >= number)
			buf = buf - number;
		L = buf;
		F(&L);
		unsigned long long mask = 0x80000000;
		unsigned int leftBit;
		for(int k = 0; k < 11; k ++) {
			leftBit = L & mask;
			L <<= 1;
			if(leftBit != 0)
				L += 1;
		}
		L ^= R;
		R = V;
	}
	unsigned long long res = (unsigned long long) L << 32;
	res += R;
	result[index] = res;
}

void createTable()
{
	short int table[8][16];
	int a = 5; 
	int c = 7;
	int m = 16;
	for(int i = 0; i < 8; i ++) {
		int t = i;
		table[i][0] = t;
		for(int j = 1; j < 16; j ++) {
			t = (a * t + c) % m;
			table[i][j] = t;
		}
	}
	for(int i = 0; i < 8; i ++ ) {
		for(int j = 0; j < 16; j ++)
			cout<<table[i][j] <<"\t";
		cout<<endl;
	}
	hipMemcpyToSymbol(HIP_SYMBOL(gTable[0]), table[0], sizeof(short int) * 16 * 8, 0, hipMemcpyHostToDevice);
}

 extern "C" void launch_gost(string fName, string kName, bool mode) 
 {
	 cout<<"Run gost algorithm\n";
	 cout<<"File name: "<<fName<<endl;
	 cout<<"Key name: "<<kName<<endl;
	 unsigned int key[8];
	 //read key
	 FILE *iKey;
	 iKey = fopen(kName.c_str(), "rb");
	 fread(&key[0], sizeof(key[0]), 8, iKey);
	 fclose(iKey);
	 //write key 
	 cout<<"Key: ";
	 for(int i = 0; i < 8; i ++)
		 cout<<key[i]<<" ";
	 cout<<endl;
	 //read data and run crypt algorithm
	 FILE *iFile;
	 FILE *oFile;
	 unsigned long long data[SIZE];
	 unsigned long long result[SIZE];
	 unsigned long long *gResult;
	 unsigned long long *gData;
	 hipMemcpyToSymbol(HIP_SYMBOL(gKey), key , sizeof(unsigned int) * 8, 0, hipMemcpyHostToDevice);
	 createTable();
	 iFile = fopen(fName.c_str(),"rb");
	 int pos = fName.find_last_of("/\\");
	 if(mode)
	     fName.insert(pos + 1, "encrypt");
	 else
		 fName.insert(pos + 1, "decrypt");
	 oFile = fopen((fName).c_str(),"wb");
	 hipMalloc((void **) &gResult, sizeof(unsigned long long) * SIZE);
	 hipMalloc((void **) &gData, sizeof(unsigned long long) * SIZE);
	 int count = 0;
	 while(count = fread(&data[0], sizeof(data[0]), SIZE, iFile)) {
     hipMemcpy(gData, data, sizeof(unsigned long long) * count, hipMemcpyHostToDevice);
	 if(mode)
	     gostEncrypt<<<dim3(10, 1, 1),dim3(count / 10, 1, 1)>>>(gData, gResult);
	 else
	     gostDeciphered<<<dim3(10, 1, 1),dim3(count / 10, 1, 1)>>>(gData, gResult);
	 hipEvent_t syncEvent;
     hipEventCreate(&syncEvent);    //������� event
     hipEventRecord(syncEvent, 0);  //���������� event
     hipEventSynchronize(syncEvent);  //�������������� event
     hipMemcpy((void *) &result, gResult, count * sizeof(unsigned long long), hipMemcpyDeviceToHost);
	 fwrite(&result[0], sizeof(result[0]), count, oFile);
     }
	 hipFree(gResult);
	 hipFree(gData);
	 fclose(iFile);
	 fclose(oFile);
	 cout<<"Finish\n";
 }

 /*-----------------------------------------------------------AES-----------------------------------------------------------------------*/

 __constant__ unsigned char gSubTable[256];
 __constant__ unsigned char gAesKey[32];
 __constant__ unsigned int gWords[60];
 
 struct node {
	 unsigned char dta[4][4];
 };

struct word {
	unsigned char data[4];
};

__device__ void subBytes(node *data) 
{
	for(int i = 0; i < 4; i ++)
		for(int j = 0; j < 4; j ++)
			data->dta[i][j] = gSubTable[data->dta[i][j]];
}

__device__ void shiftRows(node *data) 
{
	unsigned char buf;
	for(int i = 0; i < 4; i ++) {
			for(int k = 0; k < i; k ++) {
				buf = data->dta[i][0];
				data->dta[i][0] = data->dta[i][1];
				data->dta[i][1] = data->dta[i][2];
				data->dta[i][2] = data->dta[i][3];
				data->dta[i][3] = buf;
			}
	}

}

__device__ unsigned char multiply(unsigned char p, unsigned char v)
{
	unsigned char mask = 0x80;
	if(p == 1)
		return v;
	if(p == 2) {
		if((v & mask) != 0) {
			v <<= 1;
			v ^= 0x1B;
			return v;
		}
		else
			return v <<= 1;
	}
	if(p == 3) {
		unsigned char buf = v;
		if((v & mask) != 0) {
			v <<= 1;
			v ^= 0x1B;
			v ^= buf;
			return v;
		}
		else {
			v <<= 1;
			v ^= buf;
			return v;
		}
			

	}
}

__device__ void mixColumns(node *data) 
{
	unsigned char b[4][4] = { {2, 3, 1, 1},
	                          {1, 2, 3, 1},
	                          {1, 1, 2, 3},
	                          {3, 1, 1, 2}};
	unsigned char r[4][4] = {0};
	for(int k = 0; k < 4; k ++) {
	    for(int i = 0; i < 4; i ++) {
		    for(int j = 0; j < 4; j ++) {
			    r[i][k] ^= multiply(b[i][j], data->dta[j][k]);
		    }
	    }
	}

	for(int i = 0; i < 4; i ++)
		for(int j = 0; j < 4; j ++)
			data->dta[i][j] = r[i][j];

}

__device__ void addRoundKey(node *data, int r)
{
	for(int j = 0; j < 4; j ++) {
		unsigned int buf = 0;
		for(int i = 3; i >= 0; i --) {
			buf += (data->dta[i][j] << 8 * i);
		}
		buf ^= gWords[4*r + j];
		for(int i = 0; i < 4; i ++) {
			unsigned char buf2 = buf & 0xFF;
			data->dta[i][j] = buf & 0xFF;
			buf >>= 8;
		}
	}
			
}


 __global__ void aesEncrypt(node *data, node *result, int rounds)
 {
	 int index = blockDim.x * blockIdx.x + threadIdx.x;
	 addRoundKey(&data[index], 0);
	 for(int i = 1; i <= rounds; i ++) {
	     subBytes(&data[index]);
	     shiftRows(&data[index]);
		 if(i != rounds)
	     mixColumns(&data[index]);
		 addRoundKey(&data[index], i);
	 }
	 result[index] = data[index];
 }

 unsigned int rc(int n) 
 {
	 unsigned int res = 0;
	 res += ((unsigned int) pow(2.0f, n - 1) % 256);
	 return res;
 }

 unsigned int subWord(unsigned char *table, unsigned int n) 
 {
	unsigned int res = 0;
	unsigned int mask = 0xFF000000;
	for(int i = 3; i >=0; i --) {
		res <<= 8;
		unsigned short buf = table[(n & mask) >> 8 * i];
		mask >>= 8;
		res += buf;
	}
	return res;
 }

 unsigned int rotWord(unsigned int n)
 {
	 unsigned int buf = n & 0xFF;
	 n >>= 8;
	 buf <<= 24;
	 n += buf;
	 return n;
 }

 void createWordsAndTable(unsigned char *key, int keySize) 
 {
	 unsigned char table[] = {
		0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76, 
        0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0, 
        0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15, 
        0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75, 
        0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84, 
        0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf, 
        0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8, 
        0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2, 
        0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73, 
        0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb, 
        0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79, 
        0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08, 
        0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a, 
        0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e, 
        0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf, 
        0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16
    };
	hipMemcpyToSymbol(HIP_SYMBOL(gSubTable), table, sizeof(table), 0, hipMemcpyHostToDevice);
	 unsigned int r = 0;
	 int nK = keySize / 32;
	 switch(keySize) {
		 case 128: r = 10;
			 break;
		 case 192: r = 12;
			 break;
		 case 256: r = 14;
			 break;
	 }
	 unsigned int *words = new unsigned int[4 * (r + 1)];
	 for(int i = 0; i < 4 * (r + 1); i ++)
		 words[i] = 0;
	 for(int i = 0; i < nK; i ++) {
		 for(int j = 0; j < 4; j ++) 
			 words[i] += ((key[4 * i + j]) << (8 * j));
		 //cout<<"words["<<i<<"]="<<words[i]<<endl;
	 }
	 for(int i = nK; i < 4 * (r + 1); i ++) {
		 unsigned int t = words[i - 1];
		 if(i % nK == 0) 
			 t = subWord(table, rotWord(t)) ^ rc(i/nK);
		 if((nK == 8) && (i % nK == 4))
			 t = subWord(table, t);
		 cout<<"t="<<t<<endl;
		 words[i] = words[i - nK] ^ t;
	 }
	 cout<<"sizeof(words)"<<words<<endl;
	 hipMemcpyToSymbol(HIP_SYMBOL(gWords), words, sizeof(unsigned int) * (4 * (r + 1)), 0, hipMemcpyHostToDevice);
 }
 


 extern "C" void launch_aes(string fName, string kName, bool mode, int keySize) 
 {
	FILE *iKey;
	 unsigned char key[16] = { 0x2b, 0x7e, 0x15, 0x16, 0x28, 0xae, 0xd2, 0xa6, 
		 0xab, 0xf7, 0x15, 0x88, 0x09, 0xcf, 0x4f, 0x3c };
	 //read key
	 //iKey = fopen(kName.c_str(), "rb");
	 //fread(key, sizeof(unsigned char), 32, iKey);
	 //(iKey);
	 hipMemcpyToSymbol(HIP_SYMBOL(gAesKey), key, sizeof(key), 0,hipMemcpyHostToDevice);
	 FILE *iFile, *oFile;
	 struct::node data[SIZE];
	 struct::node result[SIZE];
	 struct::node *gData;
	 struct::node *gResult;
	 iFile = fopen(fName.c_str(), "rb");
	 int pos = fName.find_last_of("/\\");
	 cout<<"step\n";
	 if(mode) {
	     fName.insert(pos + 1, "encrypt");
		 createWordsAndTable(key, keySize);
	 }
	 else
		 fName.insert(pos + 1, "decrypt");
	 oFile = fopen(fName.c_str(), "wb");
	 int count = 1;
	 data[0].dta[0][0] = 0x32;
	 data[0].dta[0][1] = 0x88;
	 data[0].dta[0][2] = 0x31;
	 data[0].dta[0][3] = 0xe0;
	 data[0].dta[1][0] = 0x43;
	 data[0].dta[1][1] = 0x5a;
	 data[0].dta[1][2] = 0x31;
	 data[0].dta[1][3] = 0x37;
	 data[0].dta[2][0] = 0xf6;
	 data[0].dta[2][1] = 0x30;
	 data[0].dta[2][2] = 0x98;
	 data[0].dta[2][3] = 0x07;
	 data[0].dta[3][0] = 0xa8;
	 data[0].dta[3][1] = 0x8d;
	 data[0].dta[3][2] = 0xa2;
	 data[0].dta[3][3] = 0x34;
	 hipMalloc((void **) &gData, sizeof(node) * SIZE);
	 hipMalloc((void **) &gResult, sizeof(node) *SIZE);
	// while(count = fread(data, sizeof(data[0]), SIZE, iFile)) {
		 hipMemcpy(gData, data, sizeof(data[0]) * count, hipMemcpyHostToDevice);
		 if(mode)
			 aesEncrypt<<<dim3(1, 1 ,1), dim3(16, 1, 1)>>>(gData, gResult, 10);
		 //synchronize
		 hipEvent_t syncEvent;
		 hipEventCreate(&syncEvent);
		 hipEventRecord(syncEvent, 0);
		 hipEventSynchronize(syncEvent);
		 hipMemcpy(result, gResult, sizeof(result[0]) * count, hipMemcpyDeviceToHost);
		 fwrite(result, sizeof(result[0]), count, oFile);
	// }
	 hipFree(gData);
	 hipFree(gResult);
	 fclose(iFile);
	 fclose(oFile);
	 cout<<"Finish\n";
 }

